#include "x.h"


int main(int argc, char** argv)
{
  x_err err;

  double* ptr{nullptr};

  x_log('i', nullptr, "%s", err.msg());

  // CUDA driver API error
  hipInit(0);

  hipDevice_t device{0};
  hipDeviceGet(&device, 0);

  hipCtx_t context{nullptr};
  hipCtxCreate(&context, 0, device);

  err.set(x_err_cu, hipErrorOutOfMemory);
  x_log('e', nullptr, "%s", err.msg());

  hipCtxDestroy(context);

  // CUDA runtime API error
  err.set(x_err_cuda, hipErrorOutOfMemory);
  x_log('e', nullptr, "%s", err.msg());

  x_check(x_err_cuda, hipMalloc<double>, &ptr, sizeof(double));
  x_check(x_err_cuda, hipFree, ptr);
  ptr = nullptr;

  auto faulty = [](double** ptr, size_t size) -> x_err {
    return x_err(x_err_cuda, hipErrorOutOfMemory);
  };
  x_check(x_err_cuda, faulty, &ptr, sizeof(double));

  return 0;
}

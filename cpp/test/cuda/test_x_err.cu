#include "x.h"


int main(int argc, char** argv)
{
  x_err err;

  x_log('i', nullptr, "%s", err.msg());

  // CUDA driver API error
  hipInit(0);

  hipDevice_t device{0};
  hipDeviceGet(&device, 0);

  hipCtx_t context{nullptr};
  hipCtxCreate(&context, 0, device);

  err.set(x_err_cu, 1);
  x_log('e', nullptr, "%s", err.msg());

  hipCtxDestroy(context);

  // CUDA runtime API error
  err.set(x_err_cuda, 700);
  x_log('e', nullptr, "%s", err.msg());

  return 0;
}

#include "x.h"


int main(int argc, char** argv)
{
  x_err err;

  double* ptr{nullptr};

  err = x_malloc(&ptr, sizeof(double));
  if (err) {
    x_log('e', nullptr, "x_malloc: %s", err.msg());
    return EXIT_FAILURE;
  }

  hipPointerAttribute_t attr;
  hipError_t cerr = hipPointerGetAttributes(&attr, ptr);
  if (cerr != hipSuccess) {
    err.set(x_err_cuda, cerr);
    x_log('e', nullptr, "hipPointerGetAttributes: %s", err.msg());
    return EXIT_FAILURE;
  }

  x_log('i', nullptr, "Pointer type: %d", static_cast<int>(attr.type));

  x_free(ptr);

  return 0;
}

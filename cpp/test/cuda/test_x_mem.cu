#include "x.h"


int main(int argc, char** argv)
{
  x_err err;

  // CUDA driver API error
  hipDeviceptr_t ptr{0};
  hipInit(0);

  hipDevice_t device{0};
  hipDeviceGet(&device, 0);

  hipCtx_t context{nullptr};
  hipCtxCreate(&context, 0, device);

  err.set(x_err_cuda, hipMalloc(&ptr, sizeof(double)));
  if (err) {
    x_log('e', nullptr, "hipMemAllocHost: %s", err.msg());
  }

  x_log('i', nullptr, "ptr type: %s", x_memtype_cu(ptr));

  if (ptr != 0) {
    err.set(x_err_cuda, hipFree(ptr));
    if (err) {
      x_log('e', nullptr, "hipFree: %s", err.msg());
    }
    ptr = 0;
  }

  hipCtxDestroy(context);

  // CUDA runtime API error
  double* mem{nullptr};

  err.set(x_err_cuda, hipMalloc(&mem, sizeof(double)));
  if (err) {
    x_log('e', nullptr, "hipMalloc: %s", err.msg());
  }

  x_log('i', nullptr, "mem type: %s", x_memtype_cuda(mem));

  if (mem != nullptr) {
    err.set(x_err_cuda, hipFree(mem));
    if (err) {
      x_log('e', nullptr, "hipFree: %s", err.msg());
    }
    mem = nullptr;
  }

  return 0;
}
